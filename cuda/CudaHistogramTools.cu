#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string>
#include <stdio.h>
#include <unistd.h>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <>

#include "CudaImageTools.h"
#include "CudaHistogramTools.h"
#include "CudaErrorHelper.h"

using namespace std;

Histogram::Histogram(Image& src, int host)
    :_src(src)
{
    
    if(_src.getNumberOfValues() == 255)
    {
        _numValues = 256;
    }
    else
    {
        _numValues = _src.getNumberOfValues();
    }
    _dev_pixels = _src.getDevPixelPtr();
    _host_values = new int[_numValues];
    _host_valuesCumulative = new double[_numValues];
    _host_lookUpTable = new unsigned char[_numValues]; 

    if(host)
    {
        host_getHistogram();
    }
    else
    {
        dev_getHistogram();
    }

}

float Histogram::dev_getHistogram(dim3 blocks, dim3 threadsPerBlock)
{
    float miliseconds = 0, ms1 = 0, ms2 = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int rows = _src.getRows();
    int cols = _src.getCols();
    int channels = _src.getNumberOfChannels();
    double numPixels = rows*cols*channels;

    unsigned char* pixelPtr = (unsigned char*)_src.getHostPixelPtr();
    int* g_partialHistograms;
    int* g_partialCumulative;
    int* sums;

    //Reset histogram- and cumulative-histogram-array with 0s
    for (int i = 0; i < _numValues; i++)
    {
        _host_values[i]=0;
        _host_lookUpTable[i]=0;
        _host_valuesCumulative[i]=0;

    }

    // RGB-Images are transformed to YUV-Color space; the histogram-class only takes the y-channel (luminance) into account
    if(_src.getColorSpace()== colorSpace::rgb)
    {
        miliseconds += _src.dev_rgb2yuv(blocks, threadsPerBlock);
    }

    // Allocate device memory:
    gpuErrchk( hipMalloc((void**)& _dev_pixels, numPixels*sizeof(unsigned char)));
    gpuErrchk( hipMalloc((void**)& _dev_values, _numValues*sizeof(int)));
    gpuErrchk( hipMalloc((void**)& _dev_valuesCumulative, _numValues*sizeof(double)));    
    gpuErrchk( hipMalloc((void**)& g_partialHistograms, _numValues*blocks.x*sizeof(int)));

    // Begin benchmark
    hipEventRecord(start);

    gpuErrchk( hipMemcpy(_dev_pixels, _src.getHostPixelPtr(), numPixels*sizeof(unsigned char), hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(_dev_values, _host_values, _numValues*sizeof(int), hipMemcpyHostToDevice));
   
    // Histogram
    partialHistograms<<<blocks, threadsPerBlock, _numValues*sizeof(int)>>>(_dev_pixels, g_partialHistograms, _numValues, rows, cols, channels);
    gpuErrchk(hipGetLastError());

    globalHistogram<<<blocks, threadsPerBlock>>>(g_partialHistograms, _dev_values, _numValues, blocks.x);
    gpuErrchk(hipGetLastError());

    // Stop benchmark
    gpuErrchk(hipMemcpy(_host_values, _dev_values, _numValues*sizeof(int), hipMemcpyDeviceToHost));
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms1, start, stop);

    // Cumulative Histogram:
    // In the next Kernel, each block scans numValues/n elements, n is the size of the padded array (in the case that numValues is not a multiple of the number of blocks)
    int n = _numValues;
    int nPartial =  128;

    gpuErrchk( hipMalloc((void**)& g_partialCumulative, n*sizeof(int)));
    gpuErrchk( hipMalloc((void**)& sums, n*sizeof(int)/nPartial));

    // Begin benchmark
    hipEventRecord(start);

    gpuErrchk( hipMemcpy(_dev_valuesCumulative, _host_valuesCumulative, _numValues*sizeof(double), hipMemcpyHostToDevice));

    partialCumulativeHistograms<<<n/nPartial, nPartial/2, nPartial*sizeof(int)>>>(_dev_values, g_partialCumulative, sums, n, nPartial);
    gpuErrchk( hipGetLastError());

    auxiliaryCumulativeHistogram<<<1, n/(2*nPartial), n*sizeof(int)/nPartial>>>(sums, n/nPartial);
    gpuErrchk( hipGetLastError());
    
    globalCumulativeHistogram<<<blocks, threadsPerBlock>>>(g_partialCumulative, sums, _dev_valuesCumulative, _numValues, nPartial, rows, cols);
    gpuErrchk( hipGetLastError());

    gpuErrchk(hipMemcpy(_host_valuesCumulative, _dev_valuesCumulative, _numValues*sizeof(double), hipMemcpyDeviceToHost));
    
    // Stop benchmark
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms2, start, stop);

    hipFree(_dev_pixels);
    hipFree(_dev_values);
    hipFree(_dev_valuesCumulative);
    hipFree(g_partialCumulative);
    hipFree(sums);
    hipFree(g_partialHistograms);   
    
    miliseconds += ms1 + ms2;

    //Convert back to RGB if necessary
    miliseconds += _src.dev_yuv2rgb(blocks, threadsPerBlock);

    return miliseconds;
}

void Histogram::host_getHistogram()
{
    int rows = _src.getRows();
    int cols = _src.getCols();
    int channels = _src.getNumberOfChannels();
    double numPixels = rows*cols*channels;

    unsigned char* pixelPtr = (unsigned char*)_src.getHostPixelPtr();
    unsigned char value = 0;
 
    //Reset _host_values-array and _lookupTable with 0s
    for (int i = 0; i < _numValues; i++)
    {
        _host_values[i]=0;
        _host_lookUpTable[i]=0;
        _host_valuesCumulative[i]=0;
    }

    // In order to make the algorithm robust against multi-channel images (converted gvp- and color images)
    // only the first image channel is used for the histogram 

    // RGB-Images are transformed to YUV-Color space; the histogram-class only takes the y-channel (luminance) into account
    if(_src.getColorSpace()== colorSpace::rgb)
    {
        _src.host_rgb2yuv();
    }

    // Histogram
    for (int i = 0; i < numPixels; i+=channels)
    {
        value = pixelPtr[i];
        _host_values[value]++; 
    }

    // Cumulative histogram

    // For the cumulative histogram the number of channels is no longer relevant
    numPixels = rows*cols;
    double cdfval=0;

    for (int i = 0; i < _numValues; i++)
    {
        cdfval += (_host_values[i])/(double)numPixels;
        _host_valuesCumulative[i] = cdfval;
    }

    // Convert back to RGB if necessary
    _src.host_yuv2rgb();
}

void Histogram::display(ostream& output)
{
    int maxVal = getMax(_host_values, _numValues, _src.getColorSpace());
    
    int normValue = 0;

    for (int i = 0; i < _numValues; i++)
    {
        output << i << "\t|"; 
        normValue = (int)100*(_host_values[i]/(float)maxVal);
        for (int j = 0; j < normValue; j++)
        {
            output << '*';
        }
        output << '\n';
    }
    
}

float Histogram::dev_equalize(dim3 blocks, dim3 threadsPerBlock)
{
    float miliseconds = 0, ms1 = 0 ;
    hipEvent_t start, stop; 
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //Load the normalized color-values into a lookup table assuming a minimum value 0 and a maximum equals the number of values
    int rows = _src.getRows();
    int cols = _src.getCols();
    int channels = _src.getNumberOfChannels();
    int numPixels = rows*cols*channels;
    unsigned char* host_pixelPtr = (unsigned char*)_src.getHostPixelPtr(); 

    if(_src.getColorSpace()==colorSpace::rgb)
    {
        miliseconds += _src.dev_rgb2yuv(blocks, threadsPerBlock);
    }

    gpuErrchk(hipMalloc((void**)& _dev_lookUpTable, _numValues*sizeof(unsigned char)));
    gpuErrchk(hipMalloc((void**)& _dev_pixels, numPixels*sizeof(unsigned char)));
    gpuErrchk(hipMalloc((void**)& _dev_valuesCumulative, _numValues*sizeof(double)));

    // Beginn benchmark 
    hipEventRecord(start);

    gpuErrchk(hipMemcpy(_dev_lookUpTable, _host_lookUpTable, _numValues*sizeof(unsigned char), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(_dev_pixels, host_pixelPtr, numPixels*sizeof(unsigned char), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(_dev_valuesCumulative, _host_valuesCumulative, _numValues*sizeof(double),hipMemcpyHostToDevice));

    equalizationLookUpTable<<<1, 256, _numValues*sizeof(unsigned char)>>>(_dev_lookUpTable, _dev_valuesCumulative, _numValues);
    gpuErrchk(hipGetLastError());

    updatePixelsFromLookUp<<<blocks, threadsPerBlock>>>(_dev_pixels, _dev_lookUpTable, rows, cols, channels);
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMemcpy(_host_lookUpTable, _dev_lookUpTable, _numValues*sizeof(unsigned char), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy( host_pixelPtr, _dev_pixels, numPixels*sizeof(unsigned char), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(_host_valuesCumulative, _dev_valuesCumulative, _numValues*sizeof(double), hipMemcpyDeviceToHost));    
    
    // Stop benchmark
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms1, start, stop);
    hipFree(_dev_lookUpTable);
    hipFree(_dev_pixels);
    hipFree(_dev_valuesCumulative);

    miliseconds += ms1;
    miliseconds += dev_getHistogram(blocks, threadsPerBlock);

    //Transform the image back to RGB-Space if necessary
    miliseconds += _src.dev_yuv2rgb(blocks, threadsPerBlock);

    return miliseconds;
}

// Source: 2010_Szeleski_Computer Vision, algorithm and Applications, 3.1.4 
void Histogram::host_equalize()
{
    int rows = _src.getRows();
    int cols = _src.getCols();
    int channels = _src.getNumberOfChannels();
    int numPixels = rows*cols*channels;
    unsigned char* pixelPtr = (unsigned char*)_src.getHostPixelPtr();

    // The normalized cumulative histogram is used as a lookup-table to getHistogram the new color values
    for (int i = 0; i < _numValues; i++)
    {
        _host_lookUpTable[i] = host_clamp( _numValues*_host_valuesCumulative[i]);

    }

    if(_src.getColorSpace()==colorSpace::rgb)
    {
        _src.host_rgb2yuv();
    }
    
    for (int i = 0; i < numPixels; i+=channels)
    {
        unsigned char oldPixelVal = pixelPtr[i];
        unsigned char newPixelVal = _host_lookUpTable[oldPixelVal];
        pixelPtr[i] = newPixelVal; 
    }

    //getHistogram new Histogram
    host_getHistogram();

    //Transform the image back to RGB-Space if necessary
    _src.host_yuv2rgb();

}

float Histogram::dev_normalize(dim3 blocks, dim3 threadsPerBlock)
{
    float miliseconds=0, ms1 = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int rows = _src.getRows();
    int cols = _src.getCols();
    int channels = _src.getNumberOfChannels();
    int numPixels = rows*cols*channels;
    unsigned char* host_pixelPtr = (unsigned char*)_src.getHostPixelPtr(); 

    unsigned char maxPixel = getMax(host_pixelPtr, numPixels, _src.getColorSpace());
    unsigned char minPixel = getMin(host_pixelPtr, numPixels, _src.getColorSpace());

    if(_src.getColorSpace()==colorSpace::rgb)
    {
        miliseconds += _src.dev_rgb2yuv(blocks, threadsPerBlock);
    }

    gpuErrchk(hipMalloc((void**)& _dev_lookUpTable, _numValues*sizeof(unsigned char)));
    gpuErrchk(hipMalloc((void**)& _dev_pixels, numPixels*sizeof(unsigned char)));


    // Begin benchmark    
    hipEventRecord(start);
    
    gpuErrchk(hipMemcpy(_dev_lookUpTable, _host_lookUpTable, _numValues*sizeof(unsigned char), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(_dev_pixels, host_pixelPtr, numPixels*sizeof(unsigned char), hipMemcpyHostToDevice));
    
    normalizationLookUpTable<<<1, 256, _numValues*sizeof(unsigned char)>>>(_dev_lookUpTable,_numValues, maxPixel, minPixel);
    gpuErrchk(hipGetLastError());

    updatePixelsFromLookUp<<<blocks, threadsPerBlock>>>(_dev_pixels, _dev_lookUpTable, rows, cols, channels);
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMemcpy(_host_lookUpTable, _dev_lookUpTable, _numValues*sizeof(unsigned char), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy( host_pixelPtr, _dev_pixels, numPixels*sizeof(unsigned char), hipMemcpyDeviceToHost));

    // Stop benchmark
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms1, start, stop);

    hipFree(_dev_lookUpTable);
    hipFree(_dev_pixels); 

    miliseconds += ms1;
    
    miliseconds += dev_getHistogram(blocks, threadsPerBlock);

    miliseconds +=_src.dev_yuv2rgb(blocks, threadsPerBlock);

    return miliseconds;
}

// Normalize Histogram
void Histogram::host_normalize()
{
    
    int rows = _src.getRows();
    int cols = _src.getCols();
    int channels = _src.getNumberOfChannels();
    int numPixels = rows*cols*channels;
    unsigned char* pixelPtr = (unsigned char*)_src.getHostPixelPtr(); 

    unsigned char maxPixel = getMax(pixelPtr, numPixels, _src.getColorSpace());
    unsigned char minPixel = getMin(pixelPtr, numPixels, _src.getColorSpace());

    // Create Lookup-table
    for (int i = 0; i < _numValues; i++)
    {
        _host_lookUpTable[i] = host_clamp(_numValues*(i - minPixel)/(double)(maxPixel-minPixel));           
    }
    
    // Normalize image
    if(_src.getColorSpace()== colorSpace::rgb)
    {
        _src.host_rgb2yuv();
    }
    
    for (int i = 0; i < numPixels; i+= channels)
    {
        unsigned char oldPixelVal = pixelPtr[i];
        unsigned char newPixelVal = _host_lookUpTable[oldPixelVal];
        pixelPtr[i] = newPixelVal; 
    }

    //getHistogram new Histogram
    host_getHistogram();

    //Transform the image back to RGB-Space if necessary
    _src.host_yuv2rgb();

}

void Histogram::save(string path)
{
    //Save the histogram into a txt-file
    path += ".txt";

    ofstream dstFile(path.data());

    dstFile << "->Histogram Values:\n";

    for(int i = 0; i < _numValues; i++)
    {
        dstFile << _host_values[i] << '\n';
    }

    dstFile << "\n-> Cumulative Histogram\n";

    for (int i = 0; i < _numValues; i++)
    {
        dstFile << _host_valuesCumulative[i] << '\n';
    }

    dstFile << "\n->Histogram representation:\n";

    display(dstFile);

    
    dstFile.close ();
}

// Implement a one-dimensional version of the local-histograms-kernel proposed in https://developer.nvidia.com/blog/gpu-pro-tip-fast-histograms-using-shared-atomics-maxwell/
__global__ void partialHistograms(unsigned char* pixelPtr, int* g_partialHistograms, int numValues, int rows, int cols, int channels)
{
    // Allocate shared memory for partial histogram (one histogram per block)
    extern __shared__ int s_partialHistogram[];

    // Local (block-intern) thread index 
    int localThreadIdx = threadIdx.x;
    int localNumThreads = blockDim.x;

    // Global thread index
    int globalThreadIdx = threadIdx.x + blockIdx.x*blockDim.x;
    int globalNumThreads = blockDim.x*gridDim.x;

    // Initialize shared memory with 0-values
    for(int i = localThreadIdx; i < numValues; i+=localNumThreads)
    {
        s_partialHistogram[i]=0;
    }
    __syncthreads();

    int val;
    // Fill partial histograms with atomic operations in shared memory
    for(int i = globalThreadIdx; i<rows*cols; i += globalNumThreads)
    {
        val = pixelPtr[i*channels];
        atomicAdd(&s_partialHistogram[val], 1);
    }
    __syncthreads();
    
    // Partial histogram from s_partialHistogram in g_partialHistograms
    // The array g_partialHistograms has a size of numBlocks*numValues 
    // Point to the section of global memory corresponding to this block
    g_partialHistograms += blockIdx.x*numValues;
    for(int i = localThreadIdx; i < numValues; i+=localNumThreads)
    {
        g_partialHistograms[i] = s_partialHistogram[i];
    } 
}

// Implement partial-histograms-reduction kernel as proposed in https://developer.nvidia.com/blog/gpu-pro-tip-fast-histograms-using-shared-atomics-maxwell/
__global__ void globalHistogram(int* g_partialHistograms, int* histogram, int numValues, int numPartialHistograms)
{
    int thread = threadIdx.x + blockIdx.x*blockDim.x;
    int numThreads = blockDim.x*gridDim.x;
    int val = 0;

    // Each thread collects all the histogram-values for a certain bin and stores them into the global histogram
    for(int i = thread; i< numValues; i+=numThreads)
    {
        for(int j = 0; j < numPartialHistograms; j++)
        {
            val += g_partialHistograms[i + j*numValues];
        }

        histogram[i] =  val;
    }
}

// Algorithm proposed in "GPU Gems 3", chapter 39 (Parallel Prefix Sum (Scan) with CUDA) for parallelization of prefix sum
// The Kernel assumes that the array size is a multiple of the number of blocks, this assumption must be checked before the kernel call
__global__ void partialCumulativeHistograms(int* values, int* g_partialCumulative, int* sums, int n, int nPartial)
{
    extern __shared__ int s_partialCumulative[];

    int localThreadIdx = threadIdx.x;
    int globalThreadIdx = threadIdx.x + blockIdx.x*blockDim.x;
    int localNumThreads = blockDim.x;

    int offset = 1;
    
    // Initialize shared memory with 0-values
    for(int i = localThreadIdx; i < nPartial; i+=localNumThreads)
    {
        s_partialCumulative[i]=0;
    }
    __syncthreads();
    
    // Copy input histogram into shared memory
    for(int i= localThreadIdx; i < nPartial>>1; i+=localNumThreads )
    {

        s_partialCumulative[i*2] = values[2*globalThreadIdx];
        s_partialCumulative[i*2 + 1] = values[2*globalThreadIdx + 1];

    }
    __syncthreads();
    
    // Up-Sweep Phase of the Sum-Scan-Algorithm
    for (int d = nPartial>>1; d > 0; d >>= 1) 
    { 
        __syncthreads();   
        if (localThreadIdx < d)    
        { 
            int a = offset*(2*localThreadIdx+1)-1;     
            int b = offset*(2*localThreadIdx+2)-1;  

            s_partialCumulative[b] += s_partialCumulative[a];    
        }    
        offset *= 2; 
    } 
    __syncthreads();  

    // Clear the last element  
    if (localThreadIdx == 0) 
    { 
        sums[blockIdx.x] =  s_partialCumulative[nPartial - 1];
        g_partialCumulative[blockIdx.x*nPartial + nPartial - 1] = s_partialCumulative[nPartial - 1];
        s_partialCumulative[nPartial - 1] = 0; 
    } 
    
    // Down-Sweep Phase of the Sum-Scan-Algorithm
    for (int d = 1; d < nPartial; d *= 2)
    {      
        offset >>= 1;      
        __syncthreads();      
        if (localThreadIdx < d)      
        { 
            int a = offset*(2*localThreadIdx+1)-1;     
            int b = offset*(2*localThreadIdx+2)-1; 
             
            int t = s_partialCumulative[a]; 
            s_partialCumulative[a] = s_partialCumulative[b]; 
            s_partialCumulative[b] += t;       
        } 
    }  
    __syncthreads(); 
    
    // Write the partial cumulative sums to global memory analog to the partialHistograms-Kernel
    g_partialCumulative += blockIdx.x*nPartial;
    //for(int i = localThreadIdx; i < n/gridDim.x; i+=localNumThreads)
    for(int i = localThreadIdx; i < nPartial - 1; i+=localNumThreads)
    {

        g_partialCumulative[i] = s_partialCumulative[i+1];
    }
}

__global__ void auxiliaryCumulativeHistogram(int* sums,  int n)
{
   //Apply parallel Scan-Sum ALgorithm to the sums-array containing the sums of the partial cumulative histograms using global memory
    extern __shared__ int s_sums[];
    int localThreadIdx = threadIdx.x;
    int localNumThreads = blockDim.x;
    //int globalThreadIdx = threadIdx.x + blockIdx.x*blockDim.x;
    
    // COpy sums-array to shared memory
    for(int i = localThreadIdx; i< n>>1; i+=localNumThreads)
    {
        s_sums[2*i] = sums[2*i];
        s_sums[2*i + 1] = sums[2*i + 1];
    }
    __syncthreads();

    int offset = 1;

    // Up-Sweep Phase of the Sum-Scan-Algorithm
    for (int d = n >>1; d > 0; d >>= 1) 
    { 
        __syncthreads();
        if (localThreadIdx < d)    
        { 
            int a = offset*(2*localThreadIdx+1)-1;     
            int b = offset*(2*localThreadIdx+2)-1;  
            s_sums[b] += s_sums[a];    
        }    
        offset *= 2; 
    } 

   /// Clear the last element  
    if (localThreadIdx == 0) 
    { 
        s_sums[n-1] = 0; 
    }

    // Down-Sweep Phase of the Sum-Scan-Algorithm
    for (int d = 1; d <n; d *= 2)
    {      
        offset >>= 1;
        __syncthreads();      
        if (localThreadIdx< d)      
        { 
            int a = offset*(2*localThreadIdx+1)-1;     
            int b = offset*(2*localThreadIdx+2)-1; 
             
            int t = s_sums[a]; 
            s_sums[a] = s_sums[b]; 
            s_sums[b] += t;       
        } 
    }
    __syncthreads();

    // Copy to global memory
    for(int i=localThreadIdx; i<n; i+=localNumThreads)
    {
        sums[i] = s_sums[i]; 
    }

}

__global__ void globalCumulativeHistogram(int* g_partialCumulative, int* sums, double* _dev_valuesCumulative, int numValues, int nPartial, int rows, int cols)
{
    int globalThreadIdx = threadIdx.x + blockIdx.x*blockDim.x; 
    int globalNumThreads = gridDim.x*blockDim.x;

    for( int i = globalThreadIdx; i<numValues; i+= globalNumThreads )
    {
        _dev_valuesCumulative[i] = (g_partialCumulative[i] + sums[i/nPartial])/(double) ( rows*cols );
    }
}

// Assuming the lookup table can have a maximum of 256 values (according to its data-type) the normalization can be carried out by a single block
// Should the grid size be bigger than 1, all further blocks remain unused, as an exchange the calculation can be implemented using only shared-memory 
__global__ void normalizationLookUpTable(unsigned char* dev_lookUpTable, int numValues, unsigned char max, unsigned char min)
{
    extern __shared__ unsigned char s_lookUpTable[];

    int localThreadIdx = threadIdx.x;
    int localNumThreads = blockDim.x;

    for(int i=localThreadIdx; i<numValues; i+=localNumThreads)
    {
        s_lookUpTable[i] = dev_clamp(numValues*(i - min)/(double)(max-min));  
    }
    __syncthreads();

    for(int i=localThreadIdx; i<numValues; i+=localNumThreads)
    {
        dev_lookUpTable[i] = s_lookUpTable[i];  
    }

}

__global__ void equalizationLookUpTable(unsigned char* dev_lookUpTable, double* dev_valuesCumulative, int numValues)
{
    extern __shared__ unsigned char s_lookUpTable[];

    int localThreadIdx = threadIdx.x;
    int localNumThreads = blockDim.x;

    for(int i=localThreadIdx; i<numValues; i+=localNumThreads)
    {
        s_lookUpTable[i] = dev_clamp(numValues*dev_valuesCumulative[i]);  
    }
    __syncthreads();

    for(int i=localThreadIdx; i<numValues; i+=localNumThreads)
    {
        dev_lookUpTable[i] = s_lookUpTable[i];  
    }
}

__global__ void updatePixelsFromLookUp( unsigned char* pixelPtr, unsigned char* dev_lookUpTable, int rows, int cols, int channels)
{
    int globalThreadIdx = threadIdx.x + blockIdx.x*blockDim.x;
    int globalNumThreads = gridDim.x*blockDim.x;

    for(int i = globalThreadIdx; i<rows*cols; i+=globalNumThreads)
    {
        int j = i*channels;
        unsigned char oldPixelVal = pixelPtr[j];
        unsigned char newPixelVal = dev_lookUpTable[oldPixelVal];
        pixelPtr[j] = newPixelVal;
    }

}