#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string>
#include <stdio.h>
#include <unistd.h>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <>

#include "CudaImageTools.h"
#include "CudaHistogramTools.h"
#include "CudaErrorHelper.h"

using namespace std;

/* 
    Remark: all methods and functions with the prefix host_ are run on the cpu
            all methods and functions with the prefic dev_ are either run on the CUDA device or contain the memory allocation, copy and kernel call to run 
            algorithms on the CUDA device

            all variables with the prefix g_ refer to global memory in CUDA device
            all variables with the prefix s_ refer to shared memory in CUDA device
            
            execution time = parallel algorithm execution time on CUDA device + device-to-host copy time + host-to-device copy time
*/            
Histogram::Histogram(Image& src, int host)
    :_src(src)
{
    // By default ist the number-of-values parameter in RGB .ppm images "255"
    // For histogram calculations, this value has to be corrected to 256 => [0;255]
    if(_src.getNumberOfValues() == 255)
    {
        _numValues = 256;
    }
    else
    {
        _numValues = _src.getNumberOfValues();
    }

    _dev_pixels = _src.getDevPixelPtr();
    _host_values = new int[_numValues];
    _host_valuesCumulative = new double[_numValues];
    _host_lookUpTable = new unsigned char[_numValues]; 

    // Build histogram and cumulative histogram on the cpu or the CUDA device according to "host" parameter
    if(host)
    {
        host_getHistogram();
    }
    else
    {
        dev_getHistogram();
    }

}

// Build histogram and cumulative histogram on CUDA device
// Return the execution time
float Histogram::dev_getHistogram(dim3 blocks, dim3 threadsPerBlock)
{
    // Initialize timing variables
    float miliseconds = 0, ms1 = 0, ms2 = 0;

    // Initialize CUDA events for timing
    hipEvent_t start1, start2, stop1, stop2;
    hipEventCreate(&start1);
    hipEventCreate(&start2);
    hipEventCreate(&stop1);
    hipEventCreate(&stop2);

    // Get Image dimensions from source
    int rows = _src.getRows();
    int cols = _src.getCols();
    int channels = _src.getNumberOfChannels();
    double numPixels = rows*cols*channels;

    // Get pointer to Image pixels from source
    unsigned char* pixelPtr = (unsigned char*)_src.getHostPixelPtr();

    // Histogram is built in two steps according to https://developer.nvidia.com/blog/gpu-pro-tip-fast-histograms-using-shared-atomics-maxwell/
    // Pointer to global CUDA device memory with partial histograms
    int* g_partialHistograms;

    // Cumulative histogram is built in three steps according to "GPU Gems 3", chapter 39 (Parallel Prefix Sum (Scan) with CUDA), section 39.2.4
    // Pointer to global CUDA device memory with partial cumulative histograms 
    int* g_partialCumulative;
    // Pointer to global CUDA device memory with offsets for distributed cumulative histogram calculation
    int* sums;

    //Reset histogram- and cumulative-histogram-array with 0s
    for (int i = 0; i < _numValues; i++)
    {
        _host_values[i]=0;
        _host_lookUpTable[i]=0;
        _host_valuesCumulative[i]=0;

    }

    // RGB-Images are transformed to YCbCr-Color space; the histogram-class only takes the y-channel (luminance) into account
    if(_src.getColorSpace()== colorSpace::rgb)
    {
        miliseconds += _src.dev_rgb2yuv(blocks, threadsPerBlock);
    }

    // Allocate device memory:
    gpuErrchk( hipMalloc((void**)& _dev_pixels, numPixels*sizeof(unsigned char)));
    gpuErrchk( hipMalloc((void**)& _dev_values, _numValues*sizeof(int)));
    gpuErrchk( hipMalloc((void**)& _dev_valuesCumulative, _numValues*sizeof(double)));    
    gpuErrchk( hipMalloc((void**)& g_partialHistograms, _numValues*blocks.x*sizeof(int)));

    // Begin benchmark
    hipEventRecord(start1);

    gpuErrchk( hipMemcpy(_dev_pixels, _src.getHostPixelPtr(), numPixels*sizeof(unsigned char), hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(_dev_values, _host_values, _numValues*sizeof(int), hipMemcpyHostToDevice));
   
    // Build partial histogram
    partialHistograms<<<blocks, threadsPerBlock, _numValues*sizeof(int)>>>(_dev_pixels, g_partialHistograms, _numValues, rows, cols, channels);
    gpuErrchk(hipGetLastError());
    //  Build global histogram
    globalHistogram<<<blocks, threadsPerBlock>>>(g_partialHistograms, _dev_values, _numValues, blocks.x);
    gpuErrchk(hipGetLastError());

    // Stop benchmark
    gpuErrchk(hipMemcpy(_host_values, _dev_values, _numValues*sizeof(int), hipMemcpyDeviceToHost));
    hipEventRecord(stop1);

    hipEventSynchronize(stop1);
    hipEventElapsedTime(&ms1, start1, stop1);

    // Cumulative Histogram:
    // In the next Kernel, each block scans numValues/n elements, n is the number of grey values in the histogram
    // (in the case that numValues is not a multiple of the number of blocks)
    int n = _numValues;
    int nPartial =  128;

    // Allocate device memory
    gpuErrchk( hipMalloc((void**)& g_partialCumulative, n*sizeof(int)));
    gpuErrchk( hipMalloc((void**)& sums, n*sizeof(int)/nPartial));

    // Begin benchmark
    hipEventRecord(start2);

    gpuErrchk( hipMemcpy(_dev_valuesCumulative, _host_valuesCumulative, _numValues*sizeof(double), hipMemcpyHostToDevice));
    // Build partial cumulative histograms 
    partialCumulativeHistograms<<<n/nPartial, nPartial/2, nPartial*sizeof(int)>>>(_dev_values, g_partialCumulative, sums, n, nPartial);
    gpuErrchk( hipGetLastError());
    // Build auxiliary array with the offsets to be appliead to each of the partial cumulative histograms
    auxiliaryCumulativeHistogram<<<1, n/(2*nPartial), n*sizeof(int)/nPartial>>>(sums, n/nPartial);
    gpuErrchk( hipGetLastError());
    // join all partial cumulatives histogram to a global cumulative histogram
    globalCumulativeHistogram<<<blocks, threadsPerBlock>>>(g_partialCumulative, sums, _dev_valuesCumulative, _numValues, nPartial, rows, cols);
    gpuErrchk( hipGetLastError());

    gpuErrchk(hipMemcpy(_host_valuesCumulative, _dev_valuesCumulative, _numValues*sizeof(double), hipMemcpyDeviceToHost));
    
    // Stop benchmark
    hipEventRecord(stop2);

    hipEventSynchronize(stop2);
    hipEventElapsedTime(&ms2, start2, stop2);

    // Free allocated space on CUDA device
    hipFree(_dev_pixels);
    hipFree(_dev_values);
    hipFree(_dev_valuesCumulative);
    hipFree(g_partialCumulative);
    hipFree(sums);
    hipFree(g_partialHistograms);   
    
    miliseconds += ms1 + ms2;

    //Convert back to RGB if necessary
    miliseconds += _src.dev_yuv2rgb(blocks, threadsPerBlock);

    return miliseconds;
}

// Build histogram and cumulative histogram sequentially on the cpu
void Histogram::host_getHistogram()
{
    // Get Image dimensions from source
    int rows = _src.getRows();
    int cols = _src.getCols();
    int channels = _src.getNumberOfChannels();
    double numPixels = rows*cols*channels;
    
    // Get pointer to Image pixels from source
    unsigned char* pixelPtr = (unsigned char*)_src.getHostPixelPtr();
    unsigned char value = 0;
 
    //Reset _host_values-array and _lookupTable with 0s
    for (int i = 0; i < _numValues; i++)
    {
        _host_values[i]=0;
        _host_lookUpTable[i]=0;
        _host_valuesCumulative[i]=0;
    }

    // In order to make the algorithm robust against multi-channel images (converted gvp- and color images)
    // only the first image channel is used for the histogram 

    // RGB-Images are transformed to YUV-Color space; the histogram-class only takes the y-channel (luminance) into account
    if(_src.getColorSpace()== colorSpace::rgb)
    {
        _src.host_rgb2yuv();
    }

    // Build histogram
    for (int i = 0; i < numPixels; i+=channels)
    {
        value = pixelPtr[i];
        _host_values[value]++; 
    }

    // Build normalized cumulative histogram (values in interval [0;1])
    // For the cumulative histogram the number of channels is no longer relevant
    numPixels = rows*cols;
    double cdfval=0;

    for (int i = 0; i < _numValues; i++)
    {
        cdfval += (_host_values[i])/(double)numPixels;
        _host_valuesCumulative[i] = cdfval;
    }

    // Convert back to RGB if necessary
    _src.host_yuv2rgb();
}

// Display histogram (y-axis -> grey value; x-axes -> normalized frequency of each value)
void Histogram::display(ostream& output)
{
    int maxVal = getMax(_host_values, _numValues, _src.getColorSpace());
    
    int normValue = 0;

    for (int i = 0; i < _numValues; i++)
    {
        output << i << "\t|"; 
        normValue = (int)100*(_host_values[i]/(float)maxVal);
        for (int j = 0; j < normValue; j++)
        {
            output << '*';
        }
        output << '\n';
    }
    
}

// Equalize image on CUDA device
// Return execution time
float Histogram::dev_equalize(dim3 blocks, dim3 threadsPerBlock)
{
    // Initialize timing variables
    float miliseconds = 0, ms1 = 0;

    // Initialize CUDA events for timing
    hipEvent_t start, stop; 
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Get Image dimensions from Source
    int rows = _src.getRows();
    int cols = _src.getCols();
    int channels = _src.getNumberOfChannels();
    int numPixels = rows*cols*channels;

    // Get pointer to Image pixels
    unsigned char* host_pixelPtr = (unsigned char*)_src.getHostPixelPtr(); 

    //Convert to YCbCr color space if necessary
    if(_src.getColorSpace()==colorSpace::rgb)
    {
        miliseconds += _src.dev_rgb2yuv(blocks, threadsPerBlock);
    }

    // Allocate memory on CUDA device
    gpuErrchk(hipMalloc((void**)& _dev_lookUpTable, _numValues*sizeof(unsigned char)));
    gpuErrchk(hipMalloc((void**)& _dev_pixels, numPixels*sizeof(unsigned char)));
    gpuErrchk(hipMalloc((void**)& _dev_valuesCumulative, _numValues*sizeof(double)));

    // Beginn benchmark 
    hipEventRecord(start);

    gpuErrchk(hipMemcpy(_dev_lookUpTable, _host_lookUpTable, _numValues*sizeof(unsigned char), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(_dev_pixels, host_pixelPtr, numPixels*sizeof(unsigned char), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(_dev_valuesCumulative, _host_valuesCumulative, _numValues*sizeof(double),hipMemcpyHostToDevice));

    // The normalized cumulative histogram is used as a lookup-table to get the new color values
    equalizationLookUpTable<<<1, 256, _numValues*sizeof(unsigned char)>>>(_dev_lookUpTable, _dev_valuesCumulative, _numValues);
    gpuErrchk(hipGetLastError());
    // Update Image pixels replacing their values with the ones in the look up table
    updatePixelsFromLookUp<<<blocks, threadsPerBlock>>>(_dev_pixels, _dev_lookUpTable, rows, cols, channels);
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMemcpy(_host_lookUpTable, _dev_lookUpTable, _numValues*sizeof(unsigned char), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy( host_pixelPtr, _dev_pixels, numPixels*sizeof(unsigned char), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(_host_valuesCumulative, _dev_valuesCumulative, _numValues*sizeof(double), hipMemcpyDeviceToHost));    
    
    // Stop benchmark
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms1, start, stop);

    // Free allocated CUDA device memory
    hipFree(_dev_lookUpTable);
    hipFree(_dev_pixels);
    hipFree(_dev_valuesCumulative);

    miliseconds += ms1;
    // Get new histogram and cumulative histogram
    miliseconds += dev_getHistogram(blocks, threadsPerBlock);

    //Transform the image back to RGB-Space if necessary
    miliseconds += _src.dev_yuv2rgb(blocks, threadsPerBlock);

    return miliseconds;
}

// Source: 2010_Szeleski_Computer Vision, algorithm and Applications, 3.1.4 
void Histogram::host_equalize()
{
    //Get Image dimensions from source
    int rows = _src.getRows();
    int cols = _src.getCols();
    int channels = _src.getNumberOfChannels();
    int numPixels = rows*cols*channels;

    //get pointer to Image pixels
    unsigned char* pixelPtr = (unsigned char*)_src.getHostPixelPtr();

    // The normalized cumulative histogram is used as a lookup-table to getHistogram the new color values
    for (int i = 0; i < _numValues; i++)
    {
        _host_lookUpTable[i] = host_clamp( _numValues*_host_valuesCumulative[i]);

    }

    // Convert Image to YCbCr if necessary 
    if(_src.getColorSpace()==colorSpace::rgb)
    {
        _src.host_rgb2yuv();
    }
    
    // Update Image pixels replacing their values with the ones in the look up table
    for (int i = 0; i < numPixels; i+=channels)
    {
        unsigned char oldPixelVal = pixelPtr[i];
        unsigned char newPixelVal = _host_lookUpTable[oldPixelVal];
        pixelPtr[i] = newPixelVal; 
    }

    //Get new histogram and cumulative histogram
    host_getHistogram();

    //Transform the image back to RGB-Space if necessary
    _src.host_yuv2rgb();

}

// Normalize histogram and image on CUDA device and return execution time
float Histogram::dev_normalize(dim3 blocks, dim3 threadsPerBlock)
{
    // Initialize timing variables
    float miliseconds=0, ms1 = 0;

    // Initialize CUDA events for benchmarking
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Get Image dimensions from source
    int rows = _src.getRows();
    int cols = _src.getCols();
    int channels = _src.getNumberOfChannels();
    int numPixels = rows*cols*channels;

    // Get pointer to Image pixels
    unsigned char* host_pixelPtr = (unsigned char*)_src.getHostPixelPtr(); 

    //  Get biggest and smallest image value on image (sequentially)
    unsigned char maxPixel = getMax(host_pixelPtr, numPixels, _src.getColorSpace());
    unsigned char minPixel = getMin(host_pixelPtr, numPixels, _src.getColorSpace());

    // Convert to YCbCr if necessary
    if(_src.getColorSpace()==colorSpace::rgb)
    {
        miliseconds += _src.dev_rgb2yuv(blocks, threadsPerBlock);
    }

    // Allocate memory on CUDA device
    gpuErrchk(hipMalloc((void**)& _dev_lookUpTable, _numValues*sizeof(unsigned char)));
    gpuErrchk(hipMalloc((void**)& _dev_pixels, numPixels*sizeof(unsigned char)));


    // Begin benchmark    
    hipEventRecord(start);
    
    gpuErrchk(hipMemcpy(_dev_lookUpTable, _host_lookUpTable, _numValues*sizeof(unsigned char), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(_dev_pixels, host_pixelPtr, numPixels*sizeof(unsigned char), hipMemcpyHostToDevice));
    // Store the normalized pixel values into  lookup table
    normalizationLookUpTable<<<1, 256, _numValues*sizeof(unsigned char)>>>(_dev_lookUpTable,_numValues, maxPixel, minPixel);
    gpuErrchk(hipGetLastError());
    // Replace the pixel values in the source image with those of the lookup table
    updatePixelsFromLookUp<<<blocks, threadsPerBlock>>>(_dev_pixels, _dev_lookUpTable, rows, cols, channels);
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMemcpy(_host_lookUpTable, _dev_lookUpTable, _numValues*sizeof(unsigned char), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy( host_pixelPtr, _dev_pixels, numPixels*sizeof(unsigned char), hipMemcpyDeviceToHost));

    // Stop benchmark
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms1, start, stop);

    // Free CUDA device memory
    hipFree(_dev_lookUpTable);
    hipFree(_dev_pixels); 

    miliseconds += ms1;
    
    // Update histogram and cumulative histogram
    miliseconds += dev_getHistogram(blocks, threadsPerBlock);
    // Convert back to RGB if necessary
    miliseconds +=_src.dev_yuv2rgb(blocks, threadsPerBlock);

    return miliseconds;
}

// Normalize Histogram
void Histogram::host_normalize()
{
    // Get Image dimensions from source
    int rows = _src.getRows();
    int cols = _src.getCols();
    int channels = _src.getNumberOfChannels();
    int numPixels = rows*cols*channels;

    // Get pointer to image pixels from source
    unsigned char* pixelPtr = (unsigned char*)_src.getHostPixelPtr(); 

    //  Get biggest and smallest image value on image 
    unsigned char maxPixel = getMax(pixelPtr, numPixels, _src.getColorSpace());
    unsigned char minPixel = getMin(pixelPtr, numPixels, _src.getColorSpace());

    // Fill Lookup-table with normalized grey values
    for (int i = 0; i < _numValues; i++)
    {
        _host_lookUpTable[i] = host_clamp(_numValues*(i - minPixel)/(double)(maxPixel-minPixel));           
    }
    
    // Convert Image to YCrCb if necessary
    if(_src.getColorSpace()== colorSpace::rgb)
    {
        _src.host_rgb2yuv();
    }

    // Update pixels with the values from the lookup table
    for (int i = 0; i < numPixels; i+= channels)
    {
        unsigned char oldPixelVal = pixelPtr[i];
        unsigned char newPixelVal = _host_lookUpTable[oldPixelVal];
        pixelPtr[i] = newPixelVal; 
    }

    //getHistogram new Histogram
    host_getHistogram();

    //Transform the image back to RGB-Space if necessary
    _src.host_yuv2rgb();

}

// Save histogram and cumulative histogram as a .txt file into the give path
void Histogram::save(string path)
{
    path += ".txt";

    ofstream dstFile(path.data());

    dstFile << "->Histogram Values:\n";

    for(int i = 0; i < _numValues; i++)
    {
        dstFile << _host_values[i] << '\n';
    }

    dstFile << "\n-> Cumulative Histogram\n";

    for (int i = 0; i < _numValues; i++)
    {
        dstFile << _host_valuesCumulative[i] << '\n';
    }

    dstFile << "\n->Histogram representation:\n";

    display(dstFile);

    
    dstFile.close ();
}

// Implement a one-dimensional version of the local-histograms-kernel proposed in https://developer.nvidia.com/blog/gpu-pro-tip-fast-histograms-using-shared-atomics-maxwell/
__global__ void partialHistograms(unsigned char* pixelPtr, int* g_partialHistograms, int numValues, int rows, int cols, int channels)
{

    extern __shared__ int s_partialHistogram[];

    // Local (block-intern) thread index 
    int localThreadIdx = threadIdx.x;
    int localNumThreads = blockDim.x;

    // Global thread index
    int globalThreadIdx = threadIdx.x + blockIdx.x*blockDim.x;
    int globalNumThreads = blockDim.x*gridDim.x;

    // Initialize shared memory with 0-values
    for(int i = localThreadIdx; i < numValues; i+=localNumThreads)
    {
        s_partialHistogram[i]=0;
    }
    __syncthreads();

    int val;
    // Fill partial histograms with atomic operations in shared memory
    for(int i = globalThreadIdx; i<rows*cols; i += globalNumThreads)
    {
        val = pixelPtr[i*channels];
        atomicAdd(&s_partialHistogram[val], 1);
    }
    __syncthreads();
    
    // Partial histogram from s_partialHistogram in g_partialHistograms
    // The array g_partialHistograms has a size of numBlocks*numValues 
    // Point to the section of global memory corresponding to this block
    g_partialHistograms += blockIdx.x*numValues;
    for(int i = localThreadIdx; i < numValues; i+=localNumThreads)
    {
        g_partialHistograms[i] = s_partialHistogram[i];
    } 
}

// Implement partial-histograms-reduction kernel as proposed in https://developer.nvidia.com/blog/gpu-pro-tip-fast-histograms-using-shared-atomics-maxwell/
__global__ void globalHistogram(int* g_partialHistograms, int* histogram, int numValues, int numPartialHistograms)
{
    int thread = threadIdx.x + blockIdx.x*blockDim.x;
    int numThreads = blockDim.x*gridDim.x;
    int val = 0;

    // Each thread collects all the histogram-values for a certain bin and stores them into the global histogram
    for(int i = thread; i< numValues; i+=numThreads)
    {
        for(int j = 0; j < numPartialHistograms; j++)
        {
            val += g_partialHistograms[i + j*numValues];
        }

        histogram[i] =  val;
    }
}

// Algorithm proposed in "GPU Gems 3", chapter 39 (Parallel Prefix Sum (Scan) with CUDA) for parallelization of prefix sum
// The Kernel assumes that the array size is a multiple of the number of blocks, this assumption must be checked before the kernel call
__global__ void partialCumulativeHistograms(int* values, int* g_partialCumulative, int* sums, int n, int nPartial)
{
    extern __shared__ int s_partialCumulative[];

    // Local (block-intern) thread index 
    int localThreadIdx = threadIdx.x;
    int localNumThreads = blockDim.x;

    int globalThreadIdx = threadIdx.x + blockIdx.x*blockDim.x;

    int offset = 1;
    
    // Initialize shared memory with 0-values
    for(int i = localThreadIdx; i < nPartial; i+=localNumThreads)
    {
        s_partialCumulative[i]=0;
    }
    __syncthreads();
    
    // Copy input histogram into shared memory
    for(int i= localThreadIdx; i < nPartial>>1; i+=localNumThreads )
    {

        s_partialCumulative[i*2] = values[2*globalThreadIdx];
        s_partialCumulative[i*2 + 1] = values[2*globalThreadIdx + 1];

    }
    __syncthreads();
    
    // Up-Sweep Phase of the Sum-Scan-Algorithm
    for (int d = nPartial>>1; d > 0; d >>= 1) 
    { 
        __syncthreads();   
        if (localThreadIdx < d)    
        { 
            int a = offset*(2*localThreadIdx+1)-1;     
            int b = offset*(2*localThreadIdx+2)-1;  

            s_partialCumulative[b] += s_partialCumulative[a];    
        }    
        offset *= 2; 
    } 
    __syncthreads();  

    // Clear the last element  
    if (localThreadIdx == 0) 
    { 
        sums[blockIdx.x] =  s_partialCumulative[nPartial - 1];
        g_partialCumulative[blockIdx.x*nPartial + nPartial - 1] = s_partialCumulative[nPartial - 1];
        s_partialCumulative[nPartial - 1] = 0; 
    } 
    
    // Down-Sweep Phase of the Sum-Scan-Algorithm
    for (int d = 1; d < nPartial; d *= 2)
    {      
        offset >>= 1;      
        __syncthreads();      
        if (localThreadIdx < d)      
        { 
            int a = offset*(2*localThreadIdx+1)-1;     
            int b = offset*(2*localThreadIdx+2)-1; 
             
            int t = s_partialCumulative[a]; 
            s_partialCumulative[a] = s_partialCumulative[b]; 
            s_partialCumulative[b] += t;       
        } 
    }  
    __syncthreads(); 
    
    // Write the partial cumulative sums to global memory analog to the partialHistograms-Kernel
    g_partialCumulative += blockIdx.x*nPartial;
    //for(int i = localThreadIdx; i < n/gridDim.x; i+=localNumThreads)
    for(int i = localThreadIdx; i < nPartial - 1; i+=localNumThreads)
    {

        g_partialCumulative[i] = s_partialCumulative[i+1];
    }
}

// Build the prefix sum of the sums-array from the partialCumulativeHistograms kernel
__global__ void auxiliaryCumulativeHistogram(int* sums,  int n)
{
   //Apply parallel Scan-Sum ALgorithm to the sums-array containing the sums of the partial cumulative histograms using global memory
    extern __shared__ int s_sums[];
    int localThreadIdx = threadIdx.x;
    int localNumThreads = blockDim.x;
    //int globalThreadIdx = threadIdx.x + blockIdx.x*blockDim.x;
    
    // COpy sums-array to shared memory
    for(int i = localThreadIdx; i< n>>1; i+=localNumThreads)
    {
        s_sums[2*i] = sums[2*i];
        s_sums[2*i + 1] = sums[2*i + 1];
    }
    __syncthreads();

    int offset = 1;

    // Up-Sweep Phase of the Sum-Scan-Algorithm
    for (int d = n >>1; d > 0; d >>= 1) 
    { 
        __syncthreads();
        if (localThreadIdx < d)    
        { 
            int a = offset*(2*localThreadIdx+1)-1;     
            int b = offset*(2*localThreadIdx+2)-1;  
            s_sums[b] += s_sums[a];    
        }    
        offset *= 2; 
    } 

   /// Clear the last element  
    if (localThreadIdx == 0) 
    { 
        s_sums[n-1] = 0; 
    }

    // Down-Sweep Phase of the Sum-Scan-Algorithm
    for (int d = 1; d <n; d *= 2)
    {      
        offset >>= 1;
        __syncthreads();      
        if (localThreadIdx< d)      
        { 
            int a = offset*(2*localThreadIdx+1)-1;     
            int b = offset*(2*localThreadIdx+2)-1; 
             
            int t = s_sums[a]; 
            s_sums[a] = s_sums[b]; 
            s_sums[b] += t;       
        } 
    }
    __syncthreads();

    // Copy to global memory
    for(int i=localThreadIdx; i<n; i+=localNumThreads)
    {
        sums[i] = s_sums[i]; 
    }

}

// Use the elements of the sums-array from auxiliaryCumulativeHistogram as an offset for the partial cumulative histograms
__global__ void globalCumulativeHistogram(int* g_partialCumulative, int* sums, double* _dev_valuesCumulative, int numValues, int nPartial, int rows, int cols)
{
    int globalThreadIdx = threadIdx.x + blockIdx.x*blockDim.x; 
    int globalNumThreads = gridDim.x*blockDim.x;

    for( int i = globalThreadIdx; i<numValues; i+= globalNumThreads )
    {
        _dev_valuesCumulative[i] = (g_partialCumulative[i] + sums[i/nPartial])/(double) ( rows*cols );
    }
}

// Assuming the lookup table can have a maximum of 256 values (according to its data-type) the normalization can be carried out by a single block
// Should the grid size be bigger than 1, all further blocks remain unused, as an exchange the calculation can be implemented using only shared-memory 
__global__ void normalizationLookUpTable(unsigned char* dev_lookUpTable, int numValues, unsigned char max, unsigned char min)
{
    extern __shared__ unsigned char s_lookUpTable[];

    // Local (block-intern) thread index 
    int localThreadIdx = threadIdx.x;
    int localNumThreads = blockDim.x;

    for(int i=localThreadIdx; i<numValues; i+=localNumThreads)
    {
        s_lookUpTable[i] = dev_clamp(numValues*(i - min)/(double)(max-min));  
    }
    __syncthreads();

    for(int i=localThreadIdx; i<numValues; i+=localNumThreads)
    {
        dev_lookUpTable[i] = s_lookUpTable[i];  
    }

}

// Fill the lookup table with the values of the cumulative histogram
__global__ void equalizationLookUpTable(unsigned char* dev_lookUpTable, double* dev_valuesCumulative, int numValues)
{
    extern __shared__ unsigned char s_lookUpTable[];

    // Local (block-intern) thread index 
    int localThreadIdx = threadIdx.x;
    int localNumThreads = blockDim.x;

    for(int i=localThreadIdx; i<numValues; i+=localNumThreads)
    {
        s_lookUpTable[i] = dev_clamp(numValues*dev_valuesCumulative[i]);  
    }
    __syncthreads();

    for(int i=localThreadIdx; i<numValues; i+=localNumThreads)
    {
        dev_lookUpTable[i] = s_lookUpTable[i];  
    }
}

// Replace pixel values with the ones in a lookup table
__global__ void updatePixelsFromLookUp( unsigned char* pixelPtr, unsigned char* dev_lookUpTable, int rows, int cols, int channels)
{
    int globalThreadIdx = threadIdx.x + blockIdx.x*blockDim.x;
    int globalNumThreads = gridDim.x*blockDim.x;

    for(int i = globalThreadIdx; i<rows*cols; i+=globalNumThreads)
    {
        int j = i*channels;
        unsigned char oldPixelVal = pixelPtr[j];
        unsigned char newPixelVal = dev_lookUpTable[oldPixelVal];
        pixelPtr[j] = newPixelVal;
    }

}