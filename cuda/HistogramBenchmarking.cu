#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <unistd.h>
#include <chrono>
#include <ctime>
#include <hip/hip_runtime.h>
#include <>

#include "CudaErrorHelper.h"
#include "CudaImageTools.h"
#include "CudaHistogramTools.h"

using namespace std;

void ColorConversionPageableBenchmarking(dim3 blocks, dim3 threadsPerBlock, int cpu_test)
{
    clock_t start_t, stop_t;
    
    
    double cpu_RGB2YCbCrTime = 0;
    double cuda_RGB2YCbCrTime  = 0; 
 
    for (int i = 0; i < 21; i++)
    {   
        cout.precision(5);
        string path = "../../Benchmark/"+ std::to_string(i) +".ppm";
        Image test(path.data());
        double miliseconds_CUDA = 0;
        double miliseconds_CPU = 0;

        // CUDA-Device
        miliseconds_CUDA += test.dev_rgb2yuv(blocks,threadsPerBlock);
        miliseconds_CUDA += test.dev_yuv2rgb(blocks,threadsPerBlock);
        cuda_RGB2YCbCrTime += miliseconds_CUDA;

        if(cpu_test)
        {
            // CPU
            start_t = clock();
            test.host_rgb2yuv();
            test.host_yuv2rgb();
            stop_t = clock();
            miliseconds_CPU +=1000.0*((double)stop_t - (double)start_t)/CLOCKS_PER_SEC;
            cpu_RGB2YCbCrTime+= miliseconds_CPU; 
        }

    }

    cout <<'\t';
    cout <<cuda_RGB2YCbCrTime;
    cout <<"\t"<< cpu_RGB2YCbCrTime ;
    if( cpu_test)
    {
        double speedUp = cpu_RGB2YCbCrTime/cuda_RGB2YCbCrTime ;
        cout << '\t'<< speedUp;
    }
    cout << '\n';
}

void ColorConversionPinnedBenchmarking(dim3 blocks, dim3 threadsPerBlock, int cpu_test)
{
    clock_t start_t, stop_t;
    
    double cpu_RGB2YCbCrTime = 0;
    double cuda_RGB2YCbCrTime  = 0; 
 

    for (int i = 0; i < 21; i++)
    {   
        cout.precision(5);
        string path = "../../Benchmark/"+ std::to_string(i) +".ppm";
        Image test(path.data());
        double miliseconds_CUDA = 0;
        double miliseconds_CPU = 0;

        // CUDA-Device
        miliseconds_CUDA += test.dev_rgb2yuv_pinned(blocks,threadsPerBlock);
        miliseconds_CUDA += test.dev_yuv2rgb_pinned(blocks,threadsPerBlock);
        cuda_RGB2YCbCrTime += miliseconds_CUDA;

        if(cpu_test)
        {
            // CPU
            start_t = clock();
            test.host_rgb2yuv();
            test.host_yuv2rgb();
            stop_t = clock();
            miliseconds_CPU +=1000.0*((double)stop_t - (double)start_t)/CLOCKS_PER_SEC;
            cpu_RGB2YCbCrTime+= miliseconds_CPU; 
        }

    }

    cout <<'\t';
    cout <<cuda_RGB2YCbCrTime;
    cout <<"\t"<< cpu_RGB2YCbCrTime ;
    if( cpu_test)
    {
        double speedUp = cpu_RGB2YCbCrTime/cuda_RGB2YCbCrTime ;
        cout << '\t'<< speedUp;
    }
    cout << '\n';
}

void ColorConversionUnifiedBenchmarking(dim3 blocks, dim3 threadsPerBlock, int cpu_test)
{
    clock_t start_t, stop_t;
    
    double cpu_RGB2YCbCrTime = 0;
    double cuda_RGB2YCbCrTime  = 0; 
 
    for (int i = 0; i < 21; i++)
    {   
        cout.precision(5);
        string path = "../../Benchmark/"+ std::to_string(i) +".ppm";
        Image test(path.data());
        double miliseconds_CUDA = 0;
        double miliseconds_CPU = 0;

        // CUDA-Device
        miliseconds_CUDA += test.dev_rgb2yuv_unified(blocks,threadsPerBlock);
        miliseconds_CUDA += test.dev_yuv2rgb_unified(blocks,threadsPerBlock);
        cuda_RGB2YCbCrTime += miliseconds_CUDA;

        if(cpu_test)
        {
            // CPU
            start_t = clock();
            test.host_rgb2yuv();
            test.host_yuv2rgb();
            stop_t = clock();
            miliseconds_CPU +=1000.0*((double)stop_t - (double)start_t)/CLOCKS_PER_SEC;
            cpu_RGB2YCbCrTime+= miliseconds_CPU; 
        }

    }

    cout <<'\t';
    cout <<cuda_RGB2YCbCrTime;
    cout <<"\t"<< cpu_RGB2YCbCrTime ;
    if( cpu_test)
    {
        double speedUp = cpu_RGB2YCbCrTime/cuda_RGB2YCbCrTime ;
        cout << '\t'<< speedUp;
    }
    cout << '\n';
}

void RGB_HistogramOperationsBenchmarking(dim3 blocks, dim3 threadsPerBlock,int  cpu_test)
{
    clock_t start_t, stop_t;
    
    double cpu_histogramTime = 0;
    double cuda_histogramTime = 0; 


    for (int i = 0; i < 21; i++)
    {   
        cout.precision(5);
        string path = "../../Benchmark/"+ std::to_string(i) +".ppm";
        Image test(path.data());
        Histogram hist(test);
        double miliseconds_CUDA = 0;
	    double miliseconds_CPU = 0;

        // CUDA-Device
        miliseconds_CUDA += hist.dev_normalize(blocks,threadsPerBlock);
        miliseconds_CUDA += hist.dev_equalize(blocks,threadsPerBlock);
        cuda_histogramTime+= miliseconds_CUDA;

        if(cpu_test)
        {
            // CPU
            start_t = clock();
            hist.host_normalize();
            hist.host_equalize();
            stop_t = clock();
            miliseconds_CPU +=1000.0*((double)stop_t - (double)start_t)/CLOCKS_PER_SEC;
            cpu_histogramTime+= miliseconds_CPU;
        }
    }

    
    cout <<'\t';
    cout <<cuda_histogramTime;
    cout <<"\t"<< cpu_histogramTime ;
    if(cpu_test)
    {
        double speedUp = cpu_histogramTime/cuda_histogramTime;
        cout << '\t'<< speedUp;
    }
    cout << '\n';
    
}

void GVP_HistogramOperationsBenchmarking(dim3 blocks, dim3 threadsPerBlock, int cpu_test)
{
    clock_t start_t, stop_t;
    
    double cpu_histogramTime = 0;
    double cuda_histogramTime = 0; 

    for (int i = 0; i < 21; i++)
    {   
        cout.precision(5);
        string path = "../../Benchmark/"+ std::to_string(i) +".pgm";
        Image test(path.data());
        Histogram hist(test);
        double miliseconds_CUDA = 0;
        double miliseconds_CPU = 0;

        // CUDA-Device
        miliseconds_CUDA += hist.dev_normalize(blocks,threadsPerBlock);
        miliseconds_CUDA += hist.dev_equalize(blocks,threadsPerBlock);
        cuda_histogramTime+= miliseconds_CUDA;

        if(cpu_test)
        {
            // CPU
            start_t = clock();
            hist.host_normalize();
            hist.host_equalize();
            stop_t = clock();
            miliseconds_CPU +=1000.0*((double)stop_t - (double)start_t)/CLOCKS_PER_SEC;
            cpu_histogramTime+= miliseconds_CPU;
        }

    }

    
    cout <<'\t';
    cout <<cuda_histogramTime;
    cout <<"\t"<< cpu_histogramTime ;
    if(cpu_test)
    {
        double speedUp = cpu_histogramTime/cuda_histogramTime;
        cout << '\t'<< speedUp;
    }
    cout << '\n';
}

int main(int argc, char* argv[])
{

    //Over the command line the user can decide whether or not to benchmark against the CPUS
    int cpu_test = 0;

    if(argc > 1 )
    {
     	cpu_test = 1;
    }


    hipDeviceProp_t deviceProperties;
    gpuErrchk(hipGetDeviceProperties(&deviceProperties, 0));

    int numSM = deviceProperties.multiProcessorCount;
    int maxThreadsPerSM = deviceProperties.maxThreadsPerMultiProcessor;   

    cout<< "\nDevice: " << deviceProperties.name;
    cout<< "\nCompute Capability:\t\t" << deviceProperties.major, deviceProperties.minor;
    cout<< "\nClock Rate:\t\t\t" << deviceProperties.clockRate/1000 << " Hz";
    cout<< "\nNumber of SMs to be used:\t" << numSM;
    cout<< "\nMax Threads per SM:\t\t" << maxThreadsPerSM;
    cout<< "\nShared Memory per Block:\t" << deviceProperties.sharedMemPerBlock/1024 << " kB";
    cout<< "\nShared Memory per SM:\t\t" << deviceProperties.sharedMemPerMultiprocessor/1024 << " kB";
    cout<< "\nTotal Global Memory:\t\t" << deviceProperties.totalGlobalMem/1024/1024/1024 << " GB";
    
    cout<<"\n\nColor Conversion using Unified memory\n";
    
    cout<<"Blocks \tThreads\t";
    cout<<"GPU[ms]\tCPU[ms]\tSpeedUp(Cuda-Device with respect to CPU)\n";     
    for(int threadsPerBlock = 32; threadsPerBlock<512; threadsPerBlock*=2)
    {
        int blocks = numSM*maxThreadsPerSM/threadsPerBlock;
        cout<< blocks<<'\t' << threadsPerBlock;
        ColorConversionUnifiedBenchmarking(blocks, threadsPerBlock, cpu_test);
    }
    
    cout<<"\n\nColor Conversion using pageable memory\n";
    
    cout<<"Blocks \tThreads\t";
    cout<<"GPU[ms]\tCPU[ms]\tSpeedUp(Cuda-Device with respect to CPU)\n";     
    for(int threadsPerBlock = 32; threadsPerBlock<512; threadsPerBlock*=2)
    {
        int blocks = numSM*maxThreadsPerSM/threadsPerBlock;
        cout<< blocks<<'\t' << threadsPerBlock;
        ColorConversionPageableBenchmarking(blocks, threadsPerBlock, cpu_test);
    }

    cout<<"\nColor Conversion using pinned memory\n";
    
    cout<<"Blocks \tThreads\t";
    cout<<"GPU[ms]\tCPU[ms]\tSpeedUp(Cuda-Device with respect to CPU)\n";     
    for(int threadsPerBlock = 32; threadsPerBlock<512; threadsPerBlock*=2)
    {
        int blocks = numSM*maxThreadsPerSM/threadsPerBlock;
        cout<< blocks<<'\t' << threadsPerBlock;
        ColorConversionPinnedBenchmarking(blocks, threadsPerBlock, cpu_test);
    }

    cout<<"\nHistograms on grey value images\n";
    
    cout<<"Blocks \tThreads\t";
    cout<<"GPU[ms]\tCPU[ms]\tSpeedUp(Cuda-Device with respect to CPU)\n";   
    for(int threadsPerBlock = 32; threadsPerBlock<512; threadsPerBlock*=2)
    {
        int blocks = numSM*maxThreadsPerSM/threadsPerBlock;

        cout<< blocks<<'\t' << threadsPerBlock;
        GVP_HistogramOperationsBenchmarking(blocks, threadsPerBlock, cpu_test);     
    }

    cout<<"\nHistograms on RGB-images\n";
    
    cout<<"Blocks \tThreads\t";
    cout<<"GPU[ms]\tCPU[ms]\tSpeedUp(Cuda-Device with respect to CPU)\n";   
    for(int threadsPerBlock = 32; threadsPerBlock<512; threadsPerBlock*=2)
    {
        int blocks = numSM*maxThreadsPerSM/threadsPerBlock;

        cout<< blocks<<'\t' << threadsPerBlock;    
        RGB_HistogramOperationsBenchmarking(blocks, threadsPerBlock, cpu_test);
    }
    
    return EXIT_SUCCESS;
}
