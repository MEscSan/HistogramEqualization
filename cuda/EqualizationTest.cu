#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <unistd.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <>

#include "CudaErrorHelper.h"
#include "CudaImageTools.h"
#include "CudaHistogramTools.h"

using namespace std;

// Equalizes and Normalizes becnhmark-images using the methods and classes implemented in the project and saves the resulting images
// Users can decide whether to test he sequential (cpu) version of the algorithms 
// Users can give their own benchmark-image path
//
// Command line call:
// ./EqualizationTest [cpu_test] [inputImgPath]
// [cpu_test]: 
//      0 => algorithms run only on the gpu
//      else => algorithms run both on gpu(parallel) and cpu(sequential)
// [inputImgPath]: path to the test image
//
// Remark:  gvp...grey-value-picture (in code and comments) 

// Normalizes and equalizes an input color image both with RGB<->YCbCr conversion and processing the R-, G- and B-chanels separately
// Algorithms un sequentially on the cpu
void CPU_RGB_HistogramOperationsTest(string path)
{
    // Initialize Image object with input picture
    Image test(path.data());

    // Get R-, G- and B- channels from test image
    Image testR = test.getChannel(0);
    Image testG = test.getChannel(1);
    Image testB = test.getChannel(2);    

    // Initialize histograms from all 4 images on the cpu
    Histogram hist=Histogram(test,1);
    Histogram histR=Histogram(testR,1);
    Histogram histG=Histogram(testG,1);
    Histogram histB=Histogram(testB,1);

    // Normalize all histograms sequentially on the host machine
    hist.host_normalize();
    histR.host_normalize();
    histG.host_normalize();
    histB.host_normalize();

    // Initialize image object to contain the separately processed R-, G- and B-Channels
    Image testMultichannel(test.getRows(), test.getCols(), colorSpace::rgb, 255, fileType::ppmBin);

    // Save the processed images into the same directory as the imput image
    string savePath = path.substr(0, path.length() - 4) +"CPU_Normalized";
    test.save(savePath.data());

    savePath = path.substr(0, path.length() - 4) + "CPU_Normalized_RGB";

    // Insert separately processed R-, G- and B-Channels into testMultichannel-image and save it
    testMultichannel.setChannel(testR, 0);
    testMultichannel.setChannel(testG, 1);
    testMultichannel.setChannel(testB, 2);
    testMultichannel.save(savePath.data());
    
    // Equalize all histograms sequentially on the host machine
    hist.host_equalize();
    histR.host_equalize();
    histG.host_equalize();
    histB.host_equalize();

    // Save YCbCr-equalized image into the same directory as the imput image
    savePath = path.substr(0, path.length() - 4) + "CPU_Equalized";    
    test.save(savePath.data());

    // Insert equalized R-, G- and B-Channels into testMultichannel-image and save it
    savePath = path.substr(0, path.length() - 4) +"CPU_Equalized_RGB";
    testMultichannel.setChannel(testR, 0);
    testMultichannel.setChannel(testG, 1);
    testMultichannel.setChannel(testB, 2);
    testMultichannel.save(savePath.data());
}

// Normalizes and equalizes an input grey value picture
// ALgorithms run sequentially on the cpu
void CPU_GVP_HistogramOperationsTest(string path)
{
    // Initialize Image object with input gvp
    Image test(path.data());

    // Initialize histogram of input image on the cpu 
    Histogram hist = Histogram(test,1);
  
    // Normalize histogram sequentially on the cpu and save it into the same directory as the imput image
    hist.host_normalize();
    string savePath = path.substr(0, path.length() - 4) +"CPU_Normalized";
    test.save(savePath.data());

    // Equalize image and histogram sequentially on the cpu and save it into the same directory as the imput image
    hist.host_equalize();
    savePath = path.substr(0, path.length() - 4) + "CPU_Equalized";    
    test.save(savePath.data());
}

// Normalizes and equalizes an input color image both with RGB<->YCbCr conversion and processing the R-, G- and B-chanels separately
// Algorithms run in parallel on the gpu
void GPU_RGB_HistogramOperationsTest(string path)
{
    // Initialize image object with input picture
    Image test(path.data());
    
    // Get R-, G- and B-channels from input picture
    Image testR = test.getChannel(0);
    Image testG = test.getChannel(1);
    Image testB = test.getChannel(2);

    // Initialize image object to contain the separately processed R-, G- and B-Channels
    Image testMultichannel(test.getRows(), test.getCols(), colorSpace::rgb, 255, fileType::ppmBin);

    // Initialize histograms on the CUDA device
    Histogram hist=Histogram(test);
    Histogram histR=Histogram(testR);
    Histogram histG=Histogram(testG);
    Histogram histB=Histogram(testB);

    // Normalize histograms on the CUDA device
    hist.dev_normalize();
    histR.dev_normalize();
    histG.dev_normalize();
    histB.dev_normalize();
    
    // Save the YCbCr-normalized image into the same directory as the imput image
    string savePath = path.substr(0, path.length() - 4)+"CUDA_Normalized";
    test.save(savePath);

    // Insert normalized R-, G- and B-Channels into testMultichannel-image and save it
    savePath = path.substr(0, path.length() - 4) + "CUDA_Normalized_RGB";
    testMultichannel.setChannel(testR, 0);
    testMultichannel.setChannel(testG, 1);
    testMultichannel.setChannel(testB, 2);
    testMultichannel.save(savePath);
    
    // Equalize histograms on the CUDA device
    hist.dev_equalize();
    histR.dev_equalize();
    histG.dev_equalize();
    histB.dev_equalize();

    // Save the YCbCr-equalized image into the same directory as the imput image
    savePath = path.substr(0, path.length() - 4) +"CUDA_Equalized";    
    test.save(savePath.data());

    // Insert equalized R-, G- and B-Channels into testMultichannel-image and save it into the same directory as the imput image
    savePath = path.substr(0, path.length() - 4)  +"CUDA_Equalized_RGB";
    testMultichannel.setChannel(testR, 0);
    testMultichannel.setChannel(testG, 1);
    testMultichannel.setChannel(testB, 2);
    testMultichannel.save(savePath.data());
}

// Normalizes and equalizes an input gvp
// Algorithms run in parallel on the gpu
void GPU_GVP_HistogramOperationsTest(string path)
{
    // Initialize image object with input gvp
    Image test(path.data());

    // Initialize histogram of input picture using the CUDA device
    Histogram hist = Histogram(test);

    // Normalize histogram of input picture on CUDA device
    hist.dev_normalize();
    
    // Save normalized image into the same directory as the imput image
    string savePath = path.substr(0, path.length() - 4)+"CUDA_Normalized";
    test.save(savePath.data());
    
    // Equalize image and histogram on the CUDA device
    hist.dev_equalize();

    //Save the equalized gvp into the same directory as the imput image
    savePath = path.substr(0, path.length() - 4) +"CUDA_Equalized";    
    test.save(savePath.data());

}

int main(int argc, char* argv[] )
{
    // Initialize image path variable with default image path
    // Default test images found on Benchmark directory (see repository)
    string inputImgPath = "../../Benchmark/";
    
    // Initialize image extension variable, default test images are both RGB-Images (.ppm) and GVPs (.pgm)
    string inputImgExtension = "ppm_pgm";

    // Get path for benchmark-image from command line if given
    if(argc > 2)
    {
        inputImgPath = argv[2];
        inputImgExtension = inputImgPath.substr(inputImgPath.length() - 3, 3);
    }    
    
    // Check if the first input parameter is unequal 0, if it is, run all algorithms on the cpu
    if(argc > 1 && atoi(argv[1])!= 0)
    {
        // If the input image is an RGB-image
        if(inputImgExtension == "ppm")
        {
            CPU_RGB_HistogramOperationsTest(inputImgPath);
        }
        // If the input image is a grey-value picture
        else if(inputImgExtension == "pgm")
        {
            CPU_GVP_HistogramOperationsTest(inputImgPath);
        }
        // If no image path given, use default path
        else
        {
            for(int i = 0; i < 21; i++ )
            {
                string defaultPath = inputImgPath + std::to_string(i) +".ppm";
                CPU_RGB_HistogramOperationsTest(defaultPath);

                defaultPath = inputImgPath + std::to_string(i) +".pgm";
                CPU_GVP_HistogramOperationsTest(defaultPath);
            }
        }
    }

    // Run parallel algorithms on the gpu
    if(inputImgExtension == "ppm")
    {
        GPU_RGB_HistogramOperationsTest(inputImgPath);
    }
    else if(inputImgExtension == "pgm")
    {
        GPU_GVP_HistogramOperationsTest(inputImgPath);
    }
    else
    {
        for(int i = 0; i < 21; i++ )
        {
            string rgbDefaultPath = inputImgPath + std::to_string(i) +".ppm";
            GPU_RGB_HistogramOperationsTest(rgbDefaultPath);

            string gvpDefaultPath = inputImgPath + std::to_string(i) +".pgm";
            GPU_GVP_HistogramOperationsTest(gvpDefaultPath);
        }
    }

    return EXIT_SUCCESS;
}
