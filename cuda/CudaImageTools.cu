#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string>
#include <cstring>
#include <stdio.h>
#include <unistd.h>
#include <iostream>
#include <fstream>
#include <time.h>
#include "CudaImageTools.h"
#include "CudaErrorHelper.h"


using namespace std;

// Generates an Image-Object out of the given path. Invalid file-type leads to an empty 1x1 binary image
Image::Image(string path)
{
        load(path);
}

// Generates an Image-Object with dimensions rowsxcols, default file-type is pbmBin, default pixel-value is 0
Image::Image(int rows, int cols, colorSpace cs, int numValues, fileType type)
{
        _rows = rows;
        _cols = cols;
        _type = type;

        // Images in pbm-format have by default only one color (binary images with two possible values: 0 and 1)
        if(type == fileType::pbmASCII || type == fileType::pbmBin)
        {
                _numValues = 1;
        }
        else
        {
                _numValues = numValues;
        }

        // Grey value pictures have  
        if(cs == colorSpace::gvp)
        {
                _channels = 1;
        }
        else
        {
                _channels = 3;
        }

        _host_pixels = new  unsigned char[_rows*_cols*_channels];
        
        unsigned char* pixelPtr = _host_pixels;

        // Fill the pixels with random values
        // Initialize Random number generator
        srand(time(NULL));

        for (int i = 0; i < _rows*_cols*_channels; i++)
        {
                *pixelPtr= (char) rand()%numValues;
                pixelPtr++;
        }
        
}

// Get- and set-methods for rows, columns, number of colors, color space and file type
/*void Image::setRows(int rows)
{
        _rows = rows;
}*/

int Image::getRows()
{
        return _rows;
} 

/*void Image::setCols(int cols)
{
        _cols = cols;
}*/

int Image::getCols()
{
        return _cols;
}

/*void Image::setType(fileType type)
{
        _type = type;
}*/

fileType Image::getType()
{
        return _type;
}

/*void Image::setNumberOfValues(int numValues)
{
        if(numValues <1)
        {
                _numValues = 1;
        }
        else if(numValues > 255)
        {
                _numValues = 255;
        }
        else
        {
                _numValues = numValues;
        }
}*/

int Image::getNumberOfValues()
{
        return _numValues;
}

unsigned char* Image::getHostPixelPtr()
{
        return _host_pixels;
}

unsigned char* Image::getDevPixelPtr()
{
        return _dev_pixels;
}

colorSpace Image::getColorSpace()
{
        return _colorSpace;
}

// Returns the image-channel at position c as a gvp-Image object
Image Image::getChannel(int c)
{
        Image channelImg = Image(_rows, _cols, colorSpace::gvp, _numValues, fileType::pgmBin);

        unsigned char* dstPtr = (unsigned char*)channelImg.getHostPixelPtr();

        // c can only have values from 0 to channels-1
        c = host_clamp(c, 0, _channels-1);
        for (int i = c; i < _rows*_cols*_channels; i+=_channels)
        {
                *dstPtr = _host_pixels[i];
                dstPtr++;
        }

        return channelImg;
}

// Inserts a gvp-Image into a channel of the right dimensions
// Does nothing if the input Image-object doesn't have the right dimensions or is a color-picture
void Image::setChannel(Image channel, int c)
{
        if(channel.getCols()==_cols, channel.getRows()==_rows && channel.getColorSpace() == colorSpace::gvp)
        {
                unsigned char* srcPtr = (unsigned char*)channel.getHostPixelPtr();

                // c can only have values from 0 to channels-1
                c = host_clamp(c, 0, _channels-1);
                for (int i = c; i < _rows*_cols*_channels; i+=_channels)
                {
                        _host_pixels[i] = *srcPtr;
                        srcPtr++;
                }       
        }

}

// Returns the number of channels (1 for gvp- and 3 for color-pictures)
int Image::getNumberOfChannels()
{
        return _channels;
}

// If the path of a valid file-type given, overwrites the Image-object with the data from the given file, else does nothing
void Image::load(string path)
{
        //Open the source file
        _src = fopen(path.c_str(), "r");    

        header srcHeader = getHeader(_src);
        _rows = srcHeader.rows;
        _cols = srcHeader.cols;
        _type = srcHeader.type;
        _numValues = srcHeader.numColors;
        
        // For the storage allocation the number of channels has to be taken into account, which depends on the file type
        // ppm-files have 3 channels
        if(_type == fileType::ppmASCII || _type == fileType::ppmBin)
        {
                _host_pixels = new unsigned char[_rows*_cols*3];
                _channels = 3;
                // Color pictures (.ppm) are always rgb-pictures
                _colorSpace = colorSpace::rgb;
        }
        else
        {
                _host_pixels = new unsigned char[_rows*_cols];
                _channels = 1;
                _colorSpace = colorSpace::gvp;
        }

        // Store the pixel values into the Image-object
        imageToArray(_rows, _cols, _channels, _type, _src, _host_pixels);

        // close the source file
        fclose(_src);
}

// Save the image object according to its file type in the given path
// Image object of type "invalidType" are stored as txt-files
void Image::save(string path)
{
        if(_type == fileType::pbmASCII || _type == fileType::pbmBin)
        {
                path += ".pbm";
        }
        else if(_type == fileType::pgmASCII || _type == fileType::pgmBin)
        {
                path += ".pgm";
        }
        else if(_type == fileType::ppmASCII || _type == fileType::ppmBin)
        {
                path += ".ppm";
        }
        else
        {
                path += ".txt";
        }

        ofstream dstImg(path.data());

        // Write the header to the new file
        string header = "P";

        // .pbm files contain no information about the number of colors 
        if(_type ==  fileType::pbmASCII || _type == fileType::pbmBin)
        {      
                dstImg <<"P"<< _type << '\n' << _cols << " " << _rows << '\n';       
        }
        else
        {   
                dstImg <<"P"<< _type << '\n' << _cols << " " << _rows << '\n' << _numValues << '\n';
        }

        // Write the pixel values to the new file
        unsigned char* pixelPtr = _host_pixels;
        
        // ASCII-Files
        if(_type==pbmASCII||_type==pgmASCII||_type==ppmASCII)
        {
                for(int i = 0; i < _rows; i++)
                {
                        for (int j = 0; j < _cols*_channels; j++)
                        {
                                dstImg << (int)(*pixelPtr) << " ";
                                pixelPtr++;
                        }
                        
                       dstImg << '\n';
                }
        }
        // Binary files
        else
        {
                for(int i = 0; i < _cols*_rows*_channels; i++)
                {
                        dstImg << *pixelPtr;
                        pixelPtr++;
                }
        }
        
        
        dstImg.close ();
}

// Converts a color-image to gvp
float Image::dev_color2gvp(dim3 blocks, dim3 threadsPerBlock)
{
        // For benchmarking
        float miliseconds = 0;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Allocate Memory in CUDA-Device
        gpuErrchk( hipMalloc((void**)&_dev_pixels, _rows*_cols*3*sizeof(unsigned char)));

        // Copy pixel array to Cuda device
        gpuErrchk( hipMemcpy(_dev_pixels, _host_pixels, _rows*_cols*3*sizeof(unsigned char), hipMemcpyHostToDevice));

        // Run the color-to-gray conversion kernel
        hipEventRecord(start);
        color2gvp<<<blocks, threadsPerBlock>>>(_dev_pixels, _colorSpace, _rows, _cols);
        hipEventRecord(stop);

        // Check if the Kernel produced any errors
        gpuErrchk(hipGetLastError());        
        
        hipEventSynchronize(stop);
        hipEventElapsedTime(&miliseconds, start, stop);

        // Copy pixel array back to host
        gpuErrchk(hipMemcpy(_host_pixels, _dev_pixels, _rows*_cols*3*sizeof(unsigned char), hipMemcpyDeviceToHost));

        // Free allocated cuda-device memory
        hipFree(_dev_pixels); 



        _colorSpace = colorSpace::gvp;
        return miliseconds;
}      

// Source: https://en.wikipedia.org/wiki/YUV#Converting_between_Y%E2%80%B2UV_and_RGB
// 2010_ Szeleski_Bildanalyse -> pg. 88
// Transform according to the equation:
//      YUV = M x RGB + K
// Conversion with correction factors, actually RGB <-> YCbCr transform according to Wikipedia
// Conversion Matrix:(CHANGE with Matrix from Nixon 13.3.6.3)
//          0.299        0.587           0.114
//  M =     -0.168736    -0.331264       0.5
//          0.5          -0.418688       -0.081312
//
// Correction vector K:
//          0
//  K =    128
//         128
//
float Image::dev_rgb2yuv(dim3 blocks, dim3 threadsPerBlock)
{
        float miliseconds = 0;
        if(_colorSpace == colorSpace::rgb)
        {
                // For benchmarking
                
                hipEvent_t start, stop;
                hipEventCreate(&start);
                hipEventCreate(&stop);
                
                gpuErrchk( hipMallocManaged((void**)&_dev_pixels, _rows*_cols*3*sizeof(unsigned char)));

                gpuErrchk( hipMemcpy(_dev_pixels, _host_pixels, _rows*_cols*3*sizeof(unsigned char), hipMemcpyHostToDevice));

                hipEventRecord(start);
                rgb2yuv<<< blocks, threadsPerBlock>>>(_dev_pixels, _rows, _cols);
                hipEventRecord(stop);
                hipEventSynchronize(stop);
                hipEventElapsedTime(&miliseconds, start, stop); 

                gpuErrchk(hipGetLastError());
       
                gpuErrchk(hipMemcpy(_host_pixels, _dev_pixels, _rows*_cols*3*sizeof(unsigned char), hipMemcpyDeviceToHost));
         
                hipFree(_dev_pixels);
          
                gpuErrchk(hipEventDestroy(start));
                gpuErrchk(hipEventDestroy(stop));
                _colorSpace = colorSpace::yuv;

        } 
        return miliseconds;
}

float Image::dev_rgb2yuv_pinned(dim3 blocks, dim3 threadsPerBlock)
{
        float miliseconds = 0;
        if(_colorSpace == colorSpace::rgb)
        {

                // host pinned memory
                unsigned char* host_pixels_pinned;
                
                // allocate host pinned memory
                gpuErrchk(hipHostAlloc((void**)&host_pixels_pinned, _rows*_cols*3*sizeof(unsigned char), hipHostMallocDefault));
                
                // Copy image from pageable to pinned memory
                memcpy(host_pixels_pinned, _host_pixels, _rows*_cols*3*sizeof(unsigned char));

                hipEvent_t start, stop;
                hipEventCreate(&start);
                hipEventCreate(&stop);
                
                gpuErrchk( hipMalloc((void**)&_dev_pixels, _rows*_cols*3*sizeof(unsigned char)));

                gpuErrchk( hipMemcpy(_dev_pixels, host_pixels_pinned, _rows*_cols*3*sizeof(unsigned char), hipMemcpyHostToDevice));

                hipEventRecord(start);
                rgb2yuv<<< blocks, threadsPerBlock>>>(_dev_pixels, _rows, _cols);
                hipEventRecord(stop);                
                hipEventSynchronize(stop);
                hipEventElapsedTime(&miliseconds, start, stop);  

                gpuErrchk(hipGetLastError());         
                
                gpuErrchk(hipMemcpy(host_pixels_pinned, _dev_pixels, _rows*_cols*3*sizeof(unsigned char), hipMemcpyDeviceToHost));
  
                //Copy back pinned memory to pageable memory
                memcpy(_host_pixels, host_pixels_pinned, _rows*_cols*3*sizeof(unsigned char));

                hipFree(_dev_pixels);
                hipHostFree(host_pixels_pinned);

                gpuErrchk(hipEventDestroy(start));
                gpuErrchk(hipEventDestroy(stop));
                
                _colorSpace = colorSpace::yuv;

        } 
        return miliseconds;
}

float Image::dev_rgb2hsv(dim3 blocks, dim3 threadsPerBlock)
{
        float miliseconds = 0;

        if(_colorSpace == colorSpace::rgb)
        {
                
                hipEvent_t start, stop;
                hipEventCreate(&start);
                hipEventCreate(&stop);

                gpuErrchk( hipMalloc((void**)&_dev_pixels, _rows*_cols*3*sizeof(unsigned char)));

                gpuErrchk( hipMemcpy(_dev_pixels, _host_pixels, _rows*_cols*3*sizeof(unsigned char), hipMemcpyHostToDevice));

                hipEventRecord(start);
                rgb2hsv<<<blocks, threadsPerBlock>>>(_dev_pixels, _rows, _cols);
                hipEventRecord(stop);

                gpuErrchk(hipGetLastError());

                gpuErrchk(hipMemcpy(_host_pixels, _dev_pixels, _rows*_cols*3*sizeof(unsigned char), hipMemcpyDeviceToHost));

                hipFree(_dev_pixels);
                
                hipEventSynchronize(stop);
                hipEventElapsedTime(&miliseconds, start, stop); 

                _colorSpace = colorSpace::hsv;
        } 

        return miliseconds;
}

// Analog to RGB->YUV transform, solving the equation for the RGB-vector (values smaller than 1e-5 rounded to 0)
//      RGB = ( M^(-1) ) x (YUV - K)
//          1   ~0              1.401999
//  M^-1 =  1   -0.344136       -0.714136
//          1   1.772           ~0
//
float Image::dev_yuv2rgb(dim3 blocks, dim3 threadsPerBlock)
{       
        float miliseconds = 0;

        if(_colorSpace == colorSpace::yuv)
        {
                hipEvent_t start, stop;
                hipEventCreate(&start);
                hipEventCreate(&stop);

                gpuErrchk( hipMalloc((void**)&_dev_pixels, _rows*_cols*3*sizeof(unsigned char)));

                gpuErrchk( hipMemcpy(_dev_pixels, _host_pixels, _rows*_cols*3*sizeof(unsigned char), hipMemcpyHostToDevice));

                hipEventRecord(start);
                yuv2rgb<<<blocks, threadsPerBlock>>>(_dev_pixels, _rows, _cols);
                hipEventRecord(stop);

                gpuErrchk(hipGetLastError());

                gpuErrchk(hipMemcpy(_host_pixels, _dev_pixels, _rows*_cols*3*sizeof(unsigned char), hipMemcpyDeviceToHost));

                hipFree(_dev_pixels);
                
                hipEventSynchronize(stop);
                hipEventElapsedTime(&miliseconds, start, stop); 

                _colorSpace = colorSpace::rgb;
        }

        return miliseconds;
}

float Image::dev_yuv2rgb_pinned(dim3 blocks, dim3 threadsPerBlock)
{       
        float miliseconds = 0;

        if(_colorSpace == colorSpace::yuv)
        {
                // host pinned memory
                unsigned char* host_pixels_pinned;
                
                // allocate host pinned memory
                gpuErrchk(hipHostAlloc((void**)&host_pixels_pinned, _rows*_cols*3*sizeof(unsigned char), hipHostMallocDefault));
                
                // Copy image from pageable to pinned memory
                memcpy(host_pixels_pinned, _host_pixels, _rows*_cols*3*sizeof(unsigned char));

                hipEvent_t start, stop;
                hipEventCreate(&start);
                hipEventCreate(&stop);

                gpuErrchk( hipMalloc((void**)&_dev_pixels, _rows*_cols*3*sizeof(unsigned char)));

                gpuErrchk( hipMemcpy(_dev_pixels, host_pixels_pinned, _rows*_cols*3*sizeof(unsigned char), hipMemcpyHostToDevice));

                hipEventRecord(start);
                yuv2rgb<<<blocks, threadsPerBlock>>>(_dev_pixels, _rows, _cols);
                hipEventRecord(stop);      
                hipEventSynchronize(stop);
                hipEventElapsedTime(&miliseconds, start, stop); 

                gpuErrchk(hipGetLastError());

                gpuErrchk(hipMemcpy(host_pixels_pinned, _dev_pixels, _rows*_cols*3*sizeof(unsigned char), hipMemcpyDeviceToHost));

                memcpy(_host_pixels, host_pixels_pinned, _rows*_cols*3*sizeof(unsigned char));

                hipFree(_dev_pixels);
                hipHostFree(host_pixels_pinned);
                
                gpuErrchk(hipEventDestroy(start));
                gpuErrchk(hipEventDestroy(stop));

                _colorSpace = colorSpace::rgb;
        }

        return miliseconds;
}

// Converts a color-image to gvp
void Image::host_color2gvp()
{
        // Only make any changes if the image is either rgb or yuv
        // For rgb-Images the average ove all three channels is calculated
        // For yuv-images, the gvp image is the y-channel (the first one)
        if(_colorSpace == colorSpace::rgb)
        {
         
                int numPixels = _rows*_cols*3;
                unsigned char r = 0, g = 0, b = 0, gv =0;

                for (int i = 0; i < numPixels; i+=3)
                {
                        r = _host_pixels[i];
                        g = _host_pixels[i+1];
                        b = _host_pixels[i+2];

                        // Source: https://docs.opencv.org/3.4/de/d25/imgproc_color_conversions.html
                        gv = (unsigned char)(0.299*r + 0.587*g + 0.114*b);

                        _host_pixels[i]= gv;
                        _host_pixels[i+1] = gv;
                        _host_pixels[i+ 2] = gv;
                }
                        
        }
        else  if(_colorSpace == colorSpace::yuv)
        {
                int numPixels = _rows*_cols*3;
                unsigned char gv =0;

                for (int i = 0; i < numPixels; i+=3)
                {
                        gv = _host_pixels[i];
                        _host_pixels[i+1] = gv;
                        _host_pixels[i+ 2] = gv;
                }
        }
}

void Image::host_rgb2yuv()
{
        if(_colorSpace == colorSpace::rgb)
        {
                int numPixels = _rows*_cols*3;
                unsigned char r = 0, g = 0, b = 0, y =0, u = 0, v =0;

                for (int i = 0; i < numPixels; i+=3)
                {
                        r = _host_pixels[i];
                        g = _host_pixels[i+1];
                        b = _host_pixels[i+2];

                        // Make sure all values are in the range [0,255]
                        y = host_clamp( 0.299*r + 0.587*g + 0.114*b );
                        u = host_clamp(-0.168736*r - 0.331264*g + 0.500*b + 128);
                        v = host_clamp( 0.500*r - 0.418688*g - 0.081312*b + 128);
                        
                        _host_pixels[i]= y;
                        _host_pixels[i+1] = u;
                        _host_pixels[i+ 2] = v;
                }
    
                _colorSpace = colorSpace::yuv;
        }
}

void Image::host_yuv2rgb()
{       
        if(_colorSpace == colorSpace::yuv)
        {
                int numPixels = _rows*_cols*3;
                unsigned char r = 0, g = 0, b = 0, y =0, u = 0, v =0;

                for (int i = 0; i < numPixels; i+=3)
                {
                        y = _host_pixels[i];
                        u = _host_pixels[i+1];
                        v = _host_pixels[i+2];

                        // Make sure all values are in the range [0,255]
                        r = host_clamp(1*y + 1.401999*(v -128));
                        g = host_clamp(1*y  - 0.344136*(u -128) - 0.714136*(v -128 ));
                        b = host_clamp(1*y + 1.772*(u -128));

                        _host_pixels[i]= r;
                        _host_pixels[i+1] = g;
                        _host_pixels[i+ 2] = b;
                }

                _colorSpace = colorSpace::rgb;
        }
}

__global__ void color2gvp(unsigned char* pixels_ptr, colorSpace color, int rows, int cols)
{ 
        // Only make any changes if the image is either rgb or yuv
        // For rgb-Images the average ove all three channels is calculated
        // For yuv-images, the gvp image is the y-channel (the first one)
        if(color == colorSpace::rgb)
        {
                int numPixels = rows*cols;
                unsigned char r = 0, g = 0, b = 0, gv =0;

                for (int i = blockIdx.x*blockDim.x + threadIdx.x; i < numPixels; i+= blockDim.x*gridDim.x)
                {
                        // Variable j in order to access all three channels of a pixel i
                        int j = i*3;
                        r = pixels_ptr[j];
                        g = pixels_ptr[j+1];
                        b = pixels_ptr[j+2];

                        // Source: https://docs.opencv.org/3.4/de/d25/imgproc_color_conversions.html
                        gv = (unsigned char)(0.299*r + 0.587*g + 0.114*b);

                        pixels_ptr[j]= gv;
                        pixels_ptr[j + 1] = gv;
                        pixels_ptr[j + 2] = gv;
                }
        }
        else  if(color == colorSpace::yuv)
        {
                int numPixels = rows*cols;
                unsigned char gv =0;

               for (int i = blockIdx.x*blockDim.x + threadIdx.x; i < numPixels; i+= blockDim.x*gridDim.x)
                {
                        int j = i*3;
                        gv = pixels_ptr[j];
                        pixels_ptr[j + 1] = gv;
                        pixels_ptr[j + 2] = gv;
                }
        }
} 

__global__ void rgb2yuv(unsigned char* pixels_ptr, int rows, int cols)
{

        int numPixels = rows*cols;
        unsigned char r= 0, g = 0, b = 0;

        for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < numPixels; i+= blockDim.x*gridDim.x)
        {
                int j = 3*i;
                r = pixels_ptr[j];
                g = pixels_ptr[j+1];
                b = pixels_ptr[j+2]; 
                
                pixels_ptr[j] = dev_clamp( 0.299*r + 0.587*g + 0.114*b);
                pixels_ptr[j+1] = dev_clamp(-0.168736*r - 0.331264*g + 0.500*b+128);
                pixels_ptr[j+ 2]  = dev_clamp( 0.5*r - 0.419*g - 0.081*b +128);       

        }
}

__global__ void rgb2hsv(unsigned char* pixels_ptr, int rows, int cols)
{

        int numPixels = rows*cols;
        unsigned char r = 0, g = 0, b = 0, h =0, s = 0, v =0, max=0, min=255;

        for (int i = blockIdx.x*blockDim.x+threadIdx.x; i < numPixels; i+= blockDim.x*gridDim.x)
        {

                int j = 3*i;
                r = pixels_ptr[j];
                g = pixels_ptr[j+1];
                b = pixels_ptr[j+2];

                for(int k = 0; k<3; k++)
                {
                        unsigned char val =pixels_ptr[j+k];
                        if(val<min)
                        {
                                min=val;
                        }
                        if(val>max)
                        {
                                max=val;
                        }
                }

                v = max;
                if(v!=0)
                {
                        s = dev_clamp(255*(v - min)/(double)v);
                }

                if(v == r)
                {
                        h=dev_clamp(30*(g-b)/(double)(v - min));
                }
                else 
                if(v == g)
                {
                        h=dev_clamp(60 + 30*(b-r)/(double)(v - min));
                }
                else
                {
                        h = dev_clamp(120 + 30*(r-g)/(double)(v - min));
                }
     

                pixels_ptr[j]= h;
                pixels_ptr[j+1] = s;
                pixels_ptr[j+ 2] = v;
        }
}

__global__ void yuv2rgb(unsigned char* pixels_ptr, int rows, int cols)
{
        int numPixels = rows*cols;
        unsigned char r = 0, g = 0, b = 0, y =0, u = 0, v =0;

        for (int i = blockIdx.x*blockDim.x + threadIdx.x; i < numPixels; i+=blockDim.x*gridDim.x)
        {
                int j = i*3;
                y = pixels_ptr[j];
                u = pixels_ptr[j+1];
                v = pixels_ptr[j+2];

                // Make sure all values are in the range [0,255]
                r = dev_clamp(y + 1.401999*(v-128));
                g = dev_clamp(y  - 0.344136*(u-128) - 0.714136*(v-128));
                b = dev_clamp(y + 1.772*(u-128));
                pixels_ptr[j]= r;
                pixels_ptr[j+1] = g;
                pixels_ptr[j+ 2] = b;
        }
}


